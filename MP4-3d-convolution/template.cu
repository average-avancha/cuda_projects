#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 8
#define MASK_WIDTH 3
#define MASK_RAD 1
#define BLOCK_WIDTH TILE_WIDTH + MASK_WIDTH - MASK_RAD
//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int x_o = blockIdx.x * TILE_WIDTH + tx;
  int y_o = blockIdx.y * TILE_WIDTH + ty;
  int z_o = blockIdx.z * TILE_WIDTH + tz;
  int x_i = x_o - MASK_RAD;
  int y_i = y_o - MASK_RAD;
  int z_i = z_o - MASK_RAD;

  __shared__ float tile[BLOCK_WIDTH][BLOCK_WIDTH][BLOCK_WIDTH];

  if ((x_i >= 0) && (y_i >= 0) && (z_i >= 0) && (x_i < x_size) && (y_i < y_size) && (z_i < z_size)){
    tile[tz][ty][tx] = input[z_i * y_size * x_size + y_i * x_size + x_i];
  }
  else {
    tile[tz][ty][tx] = 0.0;
  }
  
  __syncthreads();

  if (tx < TILE_WIDTH && ty < TILE_WIDTH && tz < TILE_WIDTH && x_o < x_size && y_o < y_size && z_o < z_size){
    float Partial_value = 0;
    for (int i = 0; i < MASK_WIDTH; i++){
      for (int j = 0; j < MASK_WIDTH; j++){
        for (int k = 0; k < MASK_WIDTH; k++){
          Partial_value += M[i][j][k] * tile[tz + i][ty + j][tx + k];
        }
      }
    }
    output[z_o * y_size * x_size + y_o * x_size + x_o] = Partial_value;
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  wbTime_stop(GPU, "Doing GPU memory allocation");
  hipMalloc((void **) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength - 3) * sizeof(float));

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  wbTime_stop(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInput, &hostInput[3], (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil( (float) ((float)x_size / (float)TILE_WIDTH) ), ceil( (float)((float)y_size / (float)TILE_WIDTH) ), ceil( (float)((float)z_size / (float)TILE_WIDTH) ));
  dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);


  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  wbTime_stop(Copy, "Copying data from the GPU");
  hipMemcpy(&hostOutput[3], deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
